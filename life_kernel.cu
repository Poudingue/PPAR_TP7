
#include <hip/hip_runtime.h>

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy,
    unsigned int domain_x, unsigned int domain_y)
{
    x = (unsigned int)(x + dx) % domain_x;	// Wrap around
    y = (unsigned int)(y + dy) % domain_y;
    return source_domain[y * domain_x + x];
}

__device__ int read_sharedCell(int * source_domain, int x, int y, int dx, int dy,
    unsigned int domainBlock,
    int * global_domain, int gx, int gy, unsigned int domain_x, unsigned int domain_y)
{
    x = (unsigned int)(x + dx);	// Wrap around
    y = (unsigned int)(y + dy);
    if(x >= domainBlock || y>= domainBlock || x<0 || y<0){
		return read_cell(global_domain, gx, gy, 0, 0,
	                       domain_x, domain_y);
	}
    return source_domain[y * domain_x + x];
}

__device__ void read_neighbors(int * source, int x, int y, int dx, int dy,
    unsigned int domainBlock, int *red, int *blue,
    int * global_domain, int gx, int gy, unsigned int domain_x, unsigned int domain_y)
{
	int cells[8];
	cells[0] =read_sharedCell(source, x, y, 0 , 1 , domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	cells[1] =read_sharedCell(source, x, y, 1 , 1 , domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	cells[2] =read_sharedCell(source, x, y, 1 , 0 , domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	cells[3] =read_sharedCell(source, x, y, 1 , -1, domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	cells[4] =read_sharedCell(source, x, y, 0 , -1, domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	cells[5] =read_sharedCell(source, x, y, -1, -1, domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	cells[6] =read_sharedCell(source, x, y, -1, 0 , domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	cells[7] =read_sharedCell(source, x, y, -1, 1 , domainBlock,
	global_domain,gx,gy,domain_x,domain_y);
	
	for(int i = 0; i<8; i++){
		if(cells[i] == 1){
			(*red)++;
		}
		else if(cells[i] == 2){
			(*blue)++;
		}
	}
}


__device__ void new_value(int * source_domain, int x, int y,
	int myself, int *red, int *blue, int *value)
{
	if(((*red) + (*blue) > 3) || ((*red) + (*blue) < 2)){
		(*value) = 0;
	}else if(myself == 0 && (*red) + (*blue) == 3){
		if((*red) > (*blue)){
			(*value) = 1;
		}else{
			(*value) = 2;
		}
	}else{
		(*value) = myself;
	}
}

// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain,
    int domain_x, int domain_y)
{
	__shared__ int sharedCells[64];
	
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
    int gx = blockIdx.x * blockDim.x + threadIdx.x;
    int gy = blockIdx.y;
    
    // Read cell
    int myself = read_cell(source_domain, tx, ty, 0, 0,
	                       domain_x, domain_y);
	                       
	int domainBlock = 64;
	                       
	sharedCells[ty]=myself;
	
	__syncthreads();
	
    
    // TODO: Read the 8 neighbors and count number of blue and red
	
	int red = 0;
	int blue = 0;
	
	read_neighbors(sharedCells, tx, ty, 0, 0, domainBlock, &red, &blue,
	source_domain, gx, gy, domain_x, domain_y);
	
	// TODO: Compute new value
	
	int value= 0;
	new_value(source_domain, tx, ty, myself, &red, &blue, &value);
	
	// TODO: Write it in dest_domain
	dest_domain[ty * domain_x + tx] = value;
}

